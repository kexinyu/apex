#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
// Another possibility:
// #include <torch/all.h>

#include <assert.h>
#include <stdio.h>

#include "type_shim.h"
#include "multi_tensor_apply.cuh"

#define BLOCK_SIZE 512
#define ILP 4

// Step 1 computes the 'update' value of regular Adam optimizer.
template<typename GRAD_T, typename T, typename UPD_T>
struct LAMBStage1Functor
{
   __device__ __forceinline__ void operator()(
    int chunk_size,
    volatile int* noop_gmem,
    TensorListMetadata<5>& tl,
    const float* per_tensor_decay,
    const float beta1,
    const float beta2,
    const float beta1_correction,
    const float beta2_correction,
    const float epsilon,
    const float clipped_global_grad_norm)
  {
    // I'd like this kernel to propagate infs/nans.
    // if(*noop_gmem == 1)
    //   return;

    printf("chunk_size:%d\n", chunk_size);

    int tensor_loc = tl.block_to_tensor[blockIdx.x];
    int tensor_num = tl.start_tensor_this_launch + tensor_loc;
    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    float decay = per_tensor_decay[tensor_num];

    GRAD_T* g = (GRAD_T*)tl.addresses[0][tensor_loc];
    g += chunk_idx*chunk_size;

    T* p = (T*)tl.addresses[1][tensor_loc];
    p += chunk_idx*chunk_size;

    T* m = (T*)tl.addresses[2][tensor_loc];
    m += chunk_idx*chunk_size;

    T* v = (T*)tl.addresses[3][tensor_loc];
    v += chunk_idx*chunk_size;

    UPD_T* update = (UPD_T*)tl.addresses[4][tensor_loc];
    update += chunk_idx*chunk_size;

    n -= chunk_idx*chunk_size;

    printf("g[0]:%.8f\n", g[0]);
    printf("p[0]:%.8f\n", p[0]);
    printf("m[0]:%.8f\n", m[0]);
    printf("v[0]:%.8f\n", v[0]);
    printf("u[0]:%.8f\n", update[0]);
    printf("n:%d\n", n);

    // see note in multi_tensor_scale_kernel.cu
    for(int i_start = 0;
            i_start < n && i_start < chunk_size;
            i_start += blockDim.x*ILP)
    {
      GRAD_T r_g[ILP];
      T r_p[ILP];
      T r_m[ILP];
      T r_v[ILP];
#pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        int i = i_start + threadIdx.x + ii*blockDim.x;
        if(i < n && i < chunk_size)
        {
	  printf("branch1\n");
	  printf("g[i]:%.8f\n", g[i]);
          printf("p[i]:%.8f\n", p[i]);
          printf("m[i]:%.8f\n", m[i]);
          printf("v[i]:%.8f\n", v[i]);
	  r_g[ii] = g[i];
          r_p[ii] = p[i];
          r_m[ii] = m[i];
          r_v[ii] = v[i];
        } else {
          r_g[ii] = GRAD_T(0);
          r_p[ii] = T(0);
          r_m[ii] = T(0);
          r_v[ii] = T(0);
        }
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        T scaled_grad = r_g[ii] / clipped_global_grad_norm;
        r_m[ii] = r_m[ii] * beta1 + (1-beta1) * scaled_grad;
        r_v[ii] = r_v[ii] * beta2 + (1-beta2) * scaled_grad * scaled_grad;
        T next_m_unbiased = r_m[ii] / beta1_correction;
        T next_v_unbiased = r_v[ii] / beta2_correction;
        T denom = std::sqrt(next_v_unbiased) + epsilon;
        r_p[ii] = (next_m_unbiased/denom) + (decay*r_p[ii]);
	printf("g:%.8f,clipped:%f,m:%f,v:%f,m_unbiased:%f,v_unbiased:%f,denom:%f,p:%f\n", r_g[ii], clipped_global_grad_norm, r_m[ii], r_v[ii], next_m_unbiased, next_v_unbiased, denom, r_p[ii]);
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        int i = i_start + threadIdx.x + ii*blockDim.x;
        if(i < n && i < chunk_size)
        {
          update[i] = (UPD_T)r_p[ii];
          m[i] = r_m[ii];
          v[i] = r_v[ii];
        }
      }
    }
  }
};

void multi_tensor_lamb_stage1_cuda(
  int chunk_size,
  at::Tensor noop_flag,
  std::vector<std::vector<at::Tensor>> tensor_lists,
  at::Tensor per_tensor_decay,
  const int step,
  const float beta1,
  const float beta2,
  const float epsilon,
  const float g_grad_norm,
  const float max_global_grad_norm)
{
  using namespace at;

  std::cout << "g_grad_norm:" << g_grad_norm << ", max_global_grad_norm:" << max_global_grad_norm << std::endl;
  float clipped_global_grad_norm = g_grad_norm > max_global_grad_norm ? g_grad_norm / max_global_grad_norm : 1.0f;
  std::cout << "clipped_global_grad_norm:" << clipped_global_grad_norm << std::endl;
  float next_step = float(step+1);
  std::cout << "next_step:" << next_step << std::endl;
  float beta1_correction = 1.0f - std::pow(beta1, next_step);
  float beta2_correction = 1.0f - std::pow(beta2, next_step);
  std::cout << "beta1_correction:" << beta1_correction << ", beta2_correction:" << beta2_correction << std::endl;
  DISPATCH_FLOAT_AND_HALF(tensor_lists[0][0].scalar_type(), 0, "lamb_stage_1",
    DISPATCH_FLOAT_AND_HALF(tensor_lists[1][0].scalar_type(), 1, "lamb_stage_1",
      DISPATCH_FLOAT_AND_HALF(tensor_lists[4][0].scalar_type(), 2, "lamb_stage_1",
        multi_tensor_apply<5>(
          BLOCK_SIZE,
          chunk_size,
          noop_flag,
          tensor_lists,
          LAMBStage1Functor<scalar_t_0, scalar_t_1, scalar_t_2>(),
          per_tensor_decay.DATA_PTR<float>(),
          beta1,
          beta2,
          beta1_correction,
          beta2_correction,
          epsilon,
          clipped_global_grad_norm); )))

  AT_CUDA_CHECK(hipGetLastError());

  // AT_CUDA_CHECK(hipDeviceSynchronize());
}
